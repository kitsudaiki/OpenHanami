#include "hip/hip_runtime.h"
/**
 * @file        gpu_kernel.cu
 *
 * @author      Tobias Anker <tobias.anker@kitsunemimi.moe>
 *
 * @copyright   Apache License Version 2.0
 *
 *      Copyright 2022 Tobias Anker
 *
 *      Licensed under the Apache License, Version 2.0 (the "License");
 *      you may not use this file except in compliance with the License.
 *      You may obtain a copy of the License at
 *
 *          http://www.apache.org/licenses/LICENSE-2.0
 *
 *      Unless required by applicable law or agreed to in writing, software
 *      distributed under the License is distributed on an "AS IS" BASIS,
 *      WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *      See the License for the specific language governing permissions and
 *      limitations under the License.
 */

#include <iostream>
#include <chrono>
#include <math.h>
#include <climits>
#include <float.h>

#include <hip/hip_runtime_api.h>

#include "../../cluster/objects.h"

/**
 * @brief function for generating random-values
 *        coming from this website:
 *            https://www.reedbeta.com/blog/hash-functions-for-gpu-rendering/
 *
 * @param input seed for random value
 *
 * @return random value
 */
__device__ __forceinline__
uint32_t pcg_hash(const uint32_t input)
{
    const uint32_t state = input * 747796405u + 2891336453u;
    const uint32_t word = ((state >> ((state >> 28u) + 4u)) ^ state) * 277803737u;
    return (word >> 22u) ^ word;
}

//==================================================================================================
//==================================================================================================
//==================================================================================================

/**
 * @brief initialize a new synpase
 *
 * @param block source-neuron-block, which is only used to hold the randamo-value
 * @param synapse pointer to the synapse, which should be (re-) initialized
 * @param clusterSettings pointer to the cluster-settings
 * @param remainingW new weight for the synapse
 * @param randomValues pointer to the buffer with all randow-values
 */
__device__ __forceinline__ void
createNewSynapse(Synapse* synapse,
                 const ClusterSettings* clusterSettings,
                 const float remainingW,
                 uint32_t& randomSeed)
{
    const float randMax = static_cast<float>(RAND_MAX);
    uint32_t signRand = 0;
    const float sigNeg = 0.5f;

    // set activation-border
    synapse->border = remainingW;

    // set initial active-counter for reduction-process
    synapse->activeCounter = 5;

    // set target neuron
    randomSeed = pcg_hash(randomSeed);
    synapse->targetNeuronId = static_cast<uint16_t>(randomSeed % NEURONS_PER_NEURONBLOCK);

    randomSeed = pcg_hash(randomSeed);
    synapse->weight = (static_cast<float>(randomSeed) / randMax) / 10.0f;

    // update weight with sign
    randomSeed = pcg_hash(randomSeed);
    signRand = randomSeed % 1000;
    synapse->weight *= static_cast<float>(1.0f - (1000.0f * sigNeg > signRand) * 2);
}

/**
 * @brief process a single synapse-section
 *
 * @param synapseSection current synapse-section to process
 * @param connection pointer to the connection-object, which is related to the section
 * @param targetNeuronBlock neuron-block, which is the target for all synapses in the section
 * @param sourceNeuron pointer to source-neuron, which had triggered the section
 * @param originLocation location of the source-neuron to mark updates
 * @param clusterSettings pointer to cluster-settings
 * @param randomValues pointer to the list with all random-values
 * @param localMem pointer to shared-memory, which should be used by the processing thread
 */
template <bool doTrain>
__device__ __forceinline__ void
synapseProcessingBackward(SynapseSection* synapseSection,
                          Connection* connection,
                          NeuronBlock* targetNeuronBlock,
                          ClusterSettings* clusterSettings,
                          uint32_t& randomSeed)
{
    __shared__ float localPotential[64];
    localPotential[threadIdx.x] = connection->potential - connection->lowerBound;

    float val = 0.0f;
    uint8_t pos = 0;
    Synapse* synapse = nullptr;
    float halfPotential = 0.0f;
    const bool isAbleToCreate = connection->origin.isInput || clusterSettings->enableCreation;

    //for(uint32_t i = 0; i < SYNAPSES_PER_SYNAPSESECTION; ++i) {
    //    synapseSection->synapses[i].tempValue = 0.0f;
    //}

    // iterate over all synapses in the section
    while (pos < SYNAPSES_PER_SYNAPSESECTION && localPotential[threadIdx.x] > 0.01f) {
        synapse = &synapseSection->synapses[pos];

        if constexpr (doTrain) {
            // create new synapse if necesarry and training is active
            if (synapse->targetNeuronId == UNINIT_STATE_8) {
                createNewSynapse(synapse,
                                 clusterSettings,
                                 localPotential[threadIdx.x],
                                 randomSeed);
                clusterSettings->enableCreation = true;
            }

            // split synapse, if necessary
            if (isAbleToCreate && localPotential[threadIdx.x] < (0.5f + connection->tollerance) * synapse->border
                && localPotential[threadIdx.x] > (0.5f - connection->tollerance) * synapse->border)
            {
                synapse->border /= 1.5f;
                synapse->weight /= 1.5f;
                connection->tollerance /= 1.2f;
                clusterSettings->enableCreation = true;
            }
        }

        if (synapse->targetNeuronId != UNINIT_STATE_8) {
            // update target-neuron
            val = synapse->weight;
            if (localPotential[threadIdx.x] < synapse->border) {
                val *= ((1.0f / synapse->border) * localPotential[threadIdx.x]);
            }
            synapseSection->synapses[synapse->targetNeuronId].tempValue += val;
        }

        // update loop-counter
        halfPotential
            += static_cast<float>(pos < SYNAPSES_PER_SYNAPSESECTION / 2) * synapse->border;
        localPotential[threadIdx.x] -= synapse->border;
        ++pos;
    }

    if constexpr (doTrain) {
        if (localPotential[threadIdx.x] > 0.00001f && isAbleToCreate) {
            connection->splitValue = halfPotential;
        }
    }
}

/**
 * @brief processSynapses
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param synapseBlocks pointer to synapse-blocks in gpu-memory
 * @param connectionBlocks pointer to connection-blocks in gpu-memory
 * @param clusterSettings pointer to cluster-settingss in gpu-memory
 * @param randomValues pointer to list with random-values in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 * @param dimY number of connections-blocks in y-direction
 */
template <bool doTrain>
__global__ void
processSynapses(NeuronBlock* neuronBlocks,
                ConnectionBlock* connectionBlocks,
                uint64_t* synapseBlockLinks,
                SynapseBlock* synapseBlocks,
                ClusterSettings* clusterSettings,
                uint32_t randomeSeed)
{
    randomeSeed += (blockIdx.x * blockDim.x) + threadIdx.x;

    // init global pointers
    NeuronBlock* targetNeuronBlock = &neuronBlocks[blockIdx.x];
    ConnectionBlock* connectionBlock = &connectionBlocks[blockIdx.x];
    SynapseBlock* synapseBlock = &synapseBlocks[synapseBlockLinks[blockIdx.x]];
    Connection* connection = &connectionBlock->connections[threadIdx.x];

    if (connection->origin.blockId != UNINIT_STATE_16 && connection->potential > 0.0f) {
        synapseProcessingBackward<doTrain>(&synapseBlock->sections[threadIdx.x],
                                           connection,
                                           targetNeuronBlock,
                                           clusterSettings,
                                           randomeSeed);
    }
}

/**
 * @brief process neurons
 *
 * @param neuronBlocks pointer to neuron-blocks in gpu-memory
 * @param synapseBlocks pointer to synapse-blocks in gpu-memory
 * @param connectionBlocks pointer to connection-blocks in gpu-memory
 * @param clusterSettings pointer to cluster-settings in gpu-memory
 * @param neuronBlockPos position-offset within the neuron-block-buffer
 * @param dimY number of connections-blocks in y-direction
 * @param isOutputHexagon true, if current hexagon is an output-hexagon
 */
template <bool doTrain>
__global__ void
processNeurons(NeuronBlock* neuronBlocks,
               uint64_t* synapseBlockLinks,
               SynapseBlock* synapseBlocks,
               ClusterSettings* clusterSettings,
               const bool isOutputHexagon)
{
    // init shared memory
    __shared__ float localInputs[64];
    localInputs[threadIdx.x] = 0.0f;

    // init global pointers
    NeuronBlock* targetNeuronBlock = &neuronBlocks[blockIdx.x];
    SynapseBlock* synapseBlock = &synapseBlocks[synapseBlockLinks[blockIdx.x]];

    // copy input-values of all releaded synpase-blocks into the neurons
    for (uint32_t i = 0; i < NUMBER_OF_SYNAPSESECTION; ++i) {
        localInputs[threadIdx.x] += synapseBlock->sections[i].synapses[threadIdx.x].tempValue;
        synapseBlock->sections[i].synapses[threadIdx.x].tempValue = 0.0f;
    }

    Neuron* neuron = &targetNeuronBlock->neurons[threadIdx.x];
    neuron->input = localInputs[threadIdx.x];

    // process neuron-content
    if(isOutputHexagon == false)
    {
        neuron->potential /= clusterSettings->neuronCooldown;
        neuron->refractoryTime = neuron->refractoryTime >> 1;

        if (neuron->refractoryTime == 0) {
            neuron->potential = clusterSettings->potentialOverflow * neuron->input;
            neuron->refractoryTime = clusterSettings->refractoryTime;
        }

        neuron->potential -= neuron->border;
        neuron->active = neuron->potential > 0.0f;
        neuron->potential = static_cast<float>(neuron->active) * neuron->potential;
        neuron->input = 0.0f;
        neuron->potential = log2(neuron->potential + 1.0f);
    }
}

/**
 * @brief process all normal- and output-hexagons and train them, if wanted.
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param hexagons pointer to local hexagons
 * @param numberOfHexagons number of hexagons
 * @param neuronBlocks pointer to local neuron-block
 * @param numberOfNeuronBlocks number of neuron-blokcs
 * @param doTrain true to run a taining-process
 */
extern "C"
void
processing_CUDA(Hexagon* hexagon,
                SynapseBlock* synapseBlocks,
                const bool doTrain)
{
    hipSetDevice(hexagon->cudaPointer.deviceId);
    uint32_t randomeSeed = rand();

    if (hexagon->header.isInputHexagon) {
        return;
    }

    // copy necessary data from host to gpu
    hipMemcpy(hexagon->cudaPointer.connectionBlocks,
               &hexagon->connectionBlocks[0],
               hexagon->connectionBlocks.size() * sizeof(ConnectionBlock),
               hipMemcpyHostToDevice);

    if (doTrain)
    {
        processSynapses<true><<<hexagon->header.numberOfBlocks, NUMBER_OF_SYNAPSESECTION>>>(
            hexagon->cudaPointer.neuronBlocks,
            hexagon->cudaPointer.connectionBlocks,
            hexagon->cudaPointer.synapseBlockLinks,
            synapseBlocks,
            hexagon->cudaPointer.clusterSettings,
            randomeSeed + hexagon->header.hexagonId);

        processNeurons<true><<<hexagon->header.numberOfBlocks, NEURONS_PER_NEURONBLOCK>>>(
            hexagon->cudaPointer.neuronBlocks,
            hexagon->cudaPointer.synapseBlockLinks,
            synapseBlocks,
            hexagon->cudaPointer.clusterSettings,
            hexagon->header.isOutputHexagon);
    }
    else
    {
        processSynapses<false><<<hexagon->header.numberOfBlocks, NUMBER_OF_SYNAPSESECTION>>>(
            hexagon->cudaPointer.neuronBlocks,
            hexagon->cudaPointer.connectionBlocks,
            hexagon->cudaPointer.synapseBlockLinks,
            synapseBlocks,
            hexagon->cudaPointer.clusterSettings,
            randomeSeed + hexagon->header.hexagonId);

        processNeurons<false><<<hexagon->header.numberOfBlocks, NEURONS_PER_NEURONBLOCK>>>(
            hexagon->cudaPointer.neuronBlocks,
            hexagon->cudaPointer.synapseBlockLinks,
            synapseBlocks,
            hexagon->cudaPointer.clusterSettings,
            hexagon->header.isOutputHexagon);
    }

    // copy resulting data back to host
    hipMemcpy(&hexagon->neuronBlocks[0],
               hexagon->cudaPointer.neuronBlocks,
               hexagon->neuronBlocks.size() * sizeof(NeuronBlock),
               hipMemcpyDeviceToHost);
}
