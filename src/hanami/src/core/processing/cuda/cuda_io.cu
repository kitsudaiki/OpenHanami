﻿/**
 * @file        gpu_kernel.cu
 *
 * @author      Tobias Anker <tobias.anker@kitsunemimi.moe>
 *
 * @copyright   Apache License Version 2.0
 *
 *      Copyright 2022 Tobias Anker
 *
 *      Licensed under the Apache License, Version 2.0 (the "License");
 *      you may not use this file except in compliance with the License.
 *      You may obtain a copy of the License at
 *
 *          http://www.apache.org/licenses/LICENSE-2.0
 *
 *      Unless required by applicable law or agreed to in writing, software
 *      distributed under the License is distributed on an "AS IS" BASIS,
 *      WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *      See the License for the specific language governing permissions and
 *      limitations under the License.
 */

#include <iostream>
#include <chrono>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include "error_handling.h"
#include "../../cluster/objects.h"

/**
 * @brief initDevice_CUDA
 * @param hostSynapseBlocks
 * @param numberOfSynapseBlocks
 * @return
 */
extern "C"
SynapseBlock*
initDevice_CUDA(SynapseBlock* hostSynapseBlocks,
                const uint32_t numberOfSynapseBlocks)
{
    SynapseBlock* deviceSynapseBlocks = nullptr;

    hipMalloc(&deviceSynapseBlocks,
               numberOfSynapseBlocks * sizeof(SynapseBlock));
    hipMemcpy(deviceSynapseBlocks,
               hostSynapseBlocks,
               numberOfSynapseBlocks * sizeof(SynapseBlock),
               hipMemcpyHostToDevice);

    CHECK_LAST_CUDA_ERROR();

    return deviceSynapseBlocks;
}

/**
 * @brief initial copy of data from the host to the gpu
 *
 * @param gpuPointer pointer to the handle-object, which will store the pointer for the gpu-buffer
 * @param clusterSettings pointer to cluster-settings on host
 * @param neuronBlocks pointer to neuron-blocks on host
 * @param tempNeuronBlocks pointer to temp-values of the neuron-blocks on host
 * @param numberOfNeuronBlocks number of neuron-blocks to copy
 * @param synapseBlocks pointer to synapse-blocks on host
 * @param numberOfSynapseBlocks number of synapse-blocks to copy
 * @param hexagons pointer to hexagons to initialize their connection-blocks, if exist
 * @param numberOfHexagons number of hexagons in the cluster to init the connection-block-buffer
 */
extern "C"
void
initHexagonOnDevice_CUDA(Hexagon* hexagon,
                         ClusterSettings* clusterSettings,
                         SynapseBlock* hostSynapseBlocks,
                         SynapseBlock* deviceSynapseBlocks)
{
    hipSetDevice(hexagon->cudaPointer.deviceId);

    // copy settings to gpu
    hipMalloc(&hexagon->cudaPointer.clusterSettings, 1 * sizeof(ClusterSettings));
    hipMemcpy(hexagon->cudaPointer.clusterSettings,
               clusterSettings,
               1 * sizeof(ClusterSettings),
               hipMemcpyHostToDevice);

    if(hexagon->neuronBlocks.size() > 0) {
        hipMalloc(&hexagon->cudaPointer.neuronBlocks,
                   hexagon->neuronBlocks.size() * sizeof(NeuronBlock));

        hipMemcpy(hexagon->cudaPointer.neuronBlocks,
                   &hexagon->neuronBlocks[0],
                   hexagon->neuronBlocks.size() * sizeof(NeuronBlock),
                   hipMemcpyHostToDevice);
    }

    if(hexagon->connectionBlocks.size() > 0) {
        hipMalloc(&hexagon->cudaPointer.connectionBlocks,
                   hexagon->connectionBlocks.size() * sizeof(ConnectionBlock));
        hipMemcpy(hexagon->cudaPointer.connectionBlocks,
                   &hexagon->connectionBlocks[0],
                   hexagon->connectionBlocks.size() * sizeof(ConnectionBlock),
                   hipMemcpyHostToDevice);
    }

    if(hexagon->synapseBlockLinks.size() > 0) {
        hipMalloc(&hexagon->cudaPointer.synapseBlockLinks,
                   hexagon->synapseBlockLinks.size() * sizeof(uint64_t));
        hipMemcpy(hexagon->cudaPointer.synapseBlockLinks,
                   &hexagon->synapseBlockLinks[0],
                   hexagon->synapseBlockLinks.size() * sizeof(uint64_t),
                   hipMemcpyHostToDevice);
    }

    for(const uint64_t link : hexagon->synapseBlockLinks) {
        hipMemcpy(&deviceSynapseBlocks[link],
                   &hostSynapseBlocks[link],
                   sizeof(SynapseBlock),
                   hipMemcpyHostToDevice);
    }
}

/**
 * @brief removed all data from the gpu, which are linked in the handle-object
 *
 * @param gpuPointer handle with all pointer to free
 */
extern "C"
void
removeFromDevice_CUDA(Hexagon* hexagon,
                      SynapseBlock* deviceSynapseBlocks)
{
    hipSetDevice(hexagon->cudaPointer.deviceId);

    hipFree(hexagon->cudaPointer.clusterSettings);

    if (hexagon->cudaPointer.neuronBlocks != nullptr)
    {
        hipFree(hexagon->cudaPointer.neuronBlocks);
        hexagon->cudaPointer.neuronBlocks = nullptr;
    }

    if (hexagon->cudaPointer.connectionBlocks != nullptr)
    {
        hipFree(hexagon->cudaPointer.connectionBlocks);
        hexagon->cudaPointer.connectionBlocks = nullptr;
    }

    if (hexagon->cudaPointer.synapseBlockLinks != nullptr)
    {
        hipFree(hexagon->cudaPointer.synapseBlockLinks);
        hexagon->cudaPointer.synapseBlockLinks = nullptr;
    }

    for(const uint64_t link : hexagon->synapseBlockLinks) {
        hipFree(&deviceSynapseBlocks[link]);
    }
}

/**
 * @brief copy all data from the gpu back to the host
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param neuronBlocks pointer to neuron-blocks on host
 * @param numberOfNeuronBlocks number of neuron-blocks to copy
 * @param synapseBlocks pointer to synpase-blocks on host
 * @param numberOfSynapseBlocks number of synpase-blocks to copy
 */
extern "C"
void
copyFromGpu_CUDA(Hexagon* hexagon,
                 SynapseBlock* hostSynapseBlocks,
                 SynapseBlock* deviceSynapseBlocks)
{
    hipSetDevice(hexagon->cudaPointer.deviceId);

    hipMemcpy(&hexagon->neuronBlocks[0],
               hexagon->cudaPointer.neuronBlocks,
               hexagon->neuronBlocks.size() * sizeof(NeuronBlock),
               hipMemcpyDeviceToHost);

    hipMemcpy(&hexagon->connectionBlocks[0],
               hexagon->cudaPointer.connectionBlocks,
               hexagon->connectionBlocks.size() * sizeof(ConnectionBlock),
               hipMemcpyDeviceToHost);

    hipMemcpy(&hexagon->synapseBlockLinks[0],
               hexagon->cudaPointer.synapseBlockLinks,
               hexagon->synapseBlockLinks.size() * sizeof(uint64_t),
               hipMemcpyDeviceToHost);

    for(const uint64_t link : hexagon->synapseBlockLinks) {
        hipMemcpy(&hostSynapseBlocks[link],
                   &deviceSynapseBlocks[link],
                   sizeof(SynapseBlock),
                   hipMemcpyDeviceToHost);
    }
}

/**
 * @brief in case the cluster was resized, these changes have to be pushed to the gpu
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param neuronBlocks pointer to local buffer with neuron-blocks to update
 * @param numberOfNeuronBlocks number of neuron-blocks to update
 * @param hexagons pointer to local hexagons to access and update their connection-blocks
 * @param numberOfHexagons number of hexagons to update
 */
extern "C"
void
update_CUDA(Hexagon* hexagon,
            SynapseBlock* deviceSynapseBlocks)
{
    hipSetDevice(hexagon->cudaPointer.deviceId);

    removeFromDevice_CUDA(hexagon, deviceSynapseBlocks);

    // allocate to resized memory for the connectionblocks on gpu
    hipMalloc(&hexagon->cudaPointer.connectionBlocks,
               hexagon->connectionBlocks.size() * sizeof(ConnectionBlock));

    hipMemcpy(hexagon->cudaPointer.connectionBlocks,
               &hexagon->connectionBlocks[0],
               hexagon->connectionBlocks.size() * sizeof(ConnectionBlock),
               hipMemcpyHostToDevice);

    // allocate to resized memory for the neuronBlocks on gpu
    hipMalloc(&hexagon->cudaPointer.neuronBlocks,
               hexagon->neuronBlocks.size() * sizeof(ConnectionBlock));

    hipMemcpy(hexagon->cudaPointer.neuronBlocks,
               &hexagon->neuronBlocks[0],
               hexagon->neuronBlocks.size() * sizeof(ConnectionBlock),
               hipMemcpyHostToDevice);

    // allocate to resized memory for the synapseBlockLinks on gpu
    hipMalloc(&hexagon->cudaPointer.synapseBlockLinks,
               hexagon->synapseBlockLinks.size() * sizeof(ConnectionBlock));

    hipMemcpy(hexagon->cudaPointer.synapseBlockLinks,
               &hexagon->synapseBlockLinks[0],
               hexagon->synapseBlockLinks.size() * sizeof(ConnectionBlock),
               hipMemcpyHostToDevice);

    hexagon->wasResized = false;
}
