#include "hip/hip_runtime.h"
/**
 * @file        gpu_kernel.cu
 *
 * @author      Tobias Anker <tobias.anker@kitsunemimi.moe>
 *
 * @copyright   Apache License Version 2.0
 *
 *      Copyright 2022 Tobias Anker
 *
 *      Licensed under the Apache License, Version 2.0 (the "License");
 *      you may not use this file except in compliance with the License.
 *      You may obtain a copy of the License at
 *
 *          http://www.apache.org/licenses/LICENSE-2.0
 *
 *      Unless required by applicable law or agreed to in writing, software
 *      distributed under the License is distributed on an "AS IS" BASIS,
 *      WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *      See the License for the specific language governing permissions and
 *      limitations under the License.
 */

#include <iostream>
#include <chrono>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include "../../cluster/objects.h"

/**
 * @brief backpropagate a synapse-section
 *
 * @param section current synapse-section
 */
__device__ __forceinline__ bool
reduceSection(SynapseSection* section)
{
    Synapse* synapse;
    uint8_t exist = 0;

    for (uint8_t pos = 0; pos < SYNAPSES_PER_SYNAPSESECTION; pos++) {
        synapse = &section->synapses[pos];

        if (synapse->targetNeuronId != UNINIT_STATE_8) {
            synapse->activeCounter -= static_cast<uint8_t>(synapse->activeCounter < 10);

            // handle active-counter
            if (synapse->activeCounter == 0) {
                if (pos < SYNAPSES_PER_SYNAPSESECTION - 1) {
                    section->synapses[pos] = section->synapses[pos + 1];
                    section->synapses[pos + 1] = Synapse();
                } else {
                    section->synapses[pos] = Synapse();
                }
            }
            else {
                exist++;
            }
        }
    }

    // return true;
    return exist != 0;
}

/**
 * @brief reduce synapse, in order to limit the amount of memory
 *
 * @param connectionBlocks pointer to connection-blocks
 * @param neuronBlocks pointer to neuron-blocks
 * @param synapseBlocks pointer to synapse-blocks
 */
__global__ void
reduceConnections(ConnectionBlock* connectionBlocks,
                  NeuronBlock* neuronBlocks,
                  SynapseBlock* synapseBlocks)
{
    Neuron* sourceNeuron = nullptr;
    NeuronBlock* sourceNeuronBlock = nullptr;
    SynapseSection* synapseSection = nullptr;

    ConnectionBlock* connectionBlock = &connectionBlocks[blockIdx.x];
    SynapseConnection* connection = &connectionBlock->connections[threadIdx.x];

    if (connection->origin.blockId != UNINIT_STATE_16) {
        synapseSection = &synapseBlocks[connectionBlock->targetSynapseBlockPos].sections[threadIdx.x];
        sourceNeuronBlock = &neuronBlocks[connection->origin.blockId];
        sourceNeuron = &sourceNeuronBlock->neurons[connection->origin.neuronId];

        // if section is complete empty, then erase it
        if (reduceSection(synapseSection) == false) {
            // initialize the creation of a new section
            sourceNeuron->isNew = 1;
            sourceNeuron->newLowerBound = connection->lowerBound;

            // mark current connection as available again
            //connection->origin.blockId = UNINIT_STATE_32;
            connection->origin.neuronId = UNINIT_STATE_8;
        }
    }
}

/**
 * @brief run backpropagaion on all normal- and output-brikcs to update the weights
 *        of the synapses.
 *
 * @param gpuPointer handle with all gpu-pointer of the cluster
 * @param bricks pointer to local bricks
 * @param numberOfBricks number of bricks
 * @param neuronBlocks pointer to local neuron-blocks
 * @param numberOfNeuronBlocks number of neuron-blocks
 */
extern "C"
void
reduction_CUDA(CudaClusterPointer* gpuPointer,
               std::vector<Brick>& bricks)
{
    hipSetDevice(gpuPointer->deviceId);


    // process all bricks on gpu
    for (int32_t brickId = bricks.size() - 1; brickId >= 0; --brickId)
    {
        Brick* brick = &bricks[brickId];
        if (brick->header.isInputBrick) {
            continue;
        }

        // copy necessary data from host to gpu
        hipMemcpy(gpuPointer->brickPointer[brickId].neuronBlocks,
                   &brick->neuronBlocks[0],
                   brick->neuronBlocks.size() * sizeof(NeuronBlock),
                   hipMemcpyHostToDevice);

        reduceConnections<<<brick->header.dimX, 64>>>(
                gpuPointer->brickPointer[brickId].connectionBlocks,
                gpuPointer->brickPointer[brickId].neuronBlocks,
                gpuPointer->synapseBlocks);


        hipMemcpy(&brick->connectionBlocks[0],
                   gpuPointer->brickPointer[brickId].connectionBlocks,
                   brick->connectionBlocks.size() * sizeof(ConnectionBlock),
                   hipMemcpyDeviceToHost);

        // copy neurons back to host
        hipMemcpy(&brick->neuronBlocks[0],
                   gpuPointer->brickPointer[brickId].neuronBlocks,
                   brick->neuronBlocks.size() * sizeof(NeuronBlock),
                   hipMemcpyDeviceToHost);
    }

}
